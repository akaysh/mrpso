#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "tests.h"
#include "helper.h"
#include "gpu_pso.h"

texture<float, 2, hipReadModeElementType> texETCMatrix;

__device__ int GetDiscreteCoordT1(float val)
{
	return   floorf(val);
}

/* Unfortunately, we cannot do external calls to device code, so we have to copy this here under a DIFFERENT name(!!!)...
 * Thanks Nvidia!
 */
__device__ float CalcMakespanT(int numTasks, int numMachines, float *matching, float *scratch)
{
	int i;
	float makespan;
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int taskOffset, machineOffset;
	float matchingVal;
	float val;
	
	makespan = 0.0f;
	taskOffset = __mul24(threadID, numTasks);
	machineOffset = __mul24(threadID, numMachines);

	//Clear our scratch table
	for (i = 0; i < numTasks; i++)
		scratch[machineOffset + GetDiscreteCoordT1(matching[taskOffset + i])] = 0.0f;

	for (i = 0; i < numTasks; i++)
	{
		matchingVal = matching[taskOffset + i];

		scratch[machineOffset + GetDiscreteCoordT1(matchingVal)] += tex2D(texETCMatrix, matchingVal, (float) i);
		val = scratch[machineOffset + GetDiscreteCoordT1(matchingVal)];

		if (val > makespan)
			makespan = val;
	}	

	return makespan;
}

__global__ void TestMakespan(int numTasks, int numMachines, int numMatchings, float *matching, float *scratch, float *outVal)
{
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (threadID < numMatchings)
		outVal[threadID] = CalcMakespanT(numTasks, numMachines, matching, scratch);
}

int TestGPUMakespan()
{
	int i;
	int passed = 1;
	hipArray *cuArray;
	float *dOut, *matching, *scratch;
	float *hMatching, *hScratch;
	int numMatchings;
	int threadsPerBlock, numBlocks;
	float *cpuMakespans, *gpuMakespans;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	BuildMachineList("machines8.txt");
	BuildTaskList("tasks80.txt");
	GenerateETCMatrix();

	numMatchings = 128;
	threadsPerBlock = 64;
	numBlocks = CalcNumBlocks(numMatchings, threadsPerBlock);

	printf("\tRunning GPU Makespan Test...\n");

	srand((unsigned int) time(NULL));

	hMatching = (float *) calloc(numMatchings * GetNumTasks(), sizeof(float));
	hScratch = (float *) calloc(numMatchings * GetNumMachines(), sizeof(float));
	cpuMakespans = (float *) malloc(numMatchings * sizeof(float));
	gpuMakespans = (float *) malloc(numMatchings * sizeof(float));

	for (i = 0; i < numMatchings * GetNumTasks(); i++)
		hMatching[i] = (float) (rand() % (GetNumMachines() * 100)) / 100.0f;

	//Compute the makespans on the CPU
	for (i = 0; i < numMatchings; i++)
		cpuMakespans[i] = ComputeMakespan(&hMatching[i * GetNumTasks()], GetNumTasks());

	hipMalloc((void **)&dOut, sizeof(float) * numMatchings );
	hipMalloc((void **)&matching, sizeof(float) * numMatchings * GetNumTasks() );
	hipMalloc((void **)&scratch, sizeof(float) * numMatchings * GetNumMachines() );

	texETCMatrix.normalized = false;
	texETCMatrix.filterMode = hipFilterModePoint;
	texETCMatrix.addressMode[0] = hipAddressModeClamp;
    texETCMatrix.addressMode[1] = hipAddressModeClamp;


	hipMallocArray(&cuArray, &channelDesc, GetNumMachines(), GetNumTasks());
	hipMemcpyToArray(cuArray, 0, 0, hETCMatrix, sizeof(float)*GetNumMachines() *GetNumTasks(), hipMemcpyHostToDevice);
	hipBindTextureToArray(texETCMatrix, cuArray, channelDesc);

	hipMemcpy(matching, hMatching, sizeof(float) * numMatchings * GetNumTasks(), hipMemcpyHostToDevice);
	hipMemcpy(scratch, hScratch, sizeof(float) * numMatchings * GetNumMachines(), hipMemcpyHostToDevice);

	TestMakespan<<<numBlocks, threadsPerBlock>>>(GetNumTasks(), GetNumMachines(), numMatchings, matching, scratch, dOut);
	hipDeviceSynchronize();

	hipMemcpy(gpuMakespans, dOut, sizeof(float) * numMatchings , hipMemcpyDeviceToHost);

	for (i = 0; i < numMatchings; i++)
	{
		if (abs(gpuMakespans[i] - cpuMakespans[i]) > ACCEPTED_DELTA)
		{
			printf("\t[ERROR] - %d GPU Makespan was: %f (expected: %f)\n", i, gpuMakespans[i], cpuMakespans[i]);
			passed = 0;
		}
	}

	PrintTestResults(passed);

	free(hMatching);
	free(hScratch);
	free(cpuMakespans);
	free(gpuMakespans);
	hipFree(dOut);
	hipFree(matching);
	hipFree(scratch);
	hipFreeArray(cuArray);

	return passed;
}

void RunSwarmFunctionTests()
{
	int passed = 1;

	printf("\nStarting GPU makespan tests...\n\n");

	passed &= TestGPUMakespan();

	if (passed)
		printf("[PASSED] All makespan tests passed!\n\n");
	else
		printf("[FAILED] makespan tests failed!\n\n");
}