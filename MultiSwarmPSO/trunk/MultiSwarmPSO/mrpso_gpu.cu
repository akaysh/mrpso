#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>
#include <hip/hip_runtime.h>
#include "helper.h"
#include "hiprand.h"

texture<float, 2, hipReadModeElementType> texETCMatrix;
hipArray *cuArray;

extern __shared__ float sharedScratch[];

__device__ float CalcMakespanShared(int numTasks, int numMachines, float *matching)
{
	int i;
	float makespan;
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int taskOffset;
	float matchingVal;
	float val;
	
	makespan = 0.0f;
	taskOffset = __mul24(threadID, numTasks);

	//Clear our scratch table
	for (i = 0; i < numTasks; i++)
		sharedScratch[(int) floorf(matching[taskOffset + i])] = 0.0f;

	for (i = 0; i < numTasks; i++)
	{
		matchingVal = matching[taskOffset + i];

		sharedScratch[(int) floorf(matchingVal)] += tex2D(texETCMatrix, matchingVal, (float) i);
		val = sharedScratch[(int) floorf(matchingVal)];

		if (val > makespan)
			makespan = val;
	}		

	return makespan;
}

__device__ float CalcMakespan(int numParticles, int numTasks, int numMachines, float *matching, float *scratch)
{
	int i;
	int swarmOffset, scratchOffset;
	float makespan;
	float matchingVal;
	float val;
	
	//The position values that this thread needs to retrieve are located at its swarm offset (mySwarm * numParticles * numTasks)
	//and we add i * numParticles as the offset at each point in the for loop.
	swarmOffset = blockIdx.x * numParticles * numTasks;
	scratchOffset = blockIdx.x * numParticles * numMachines;

	makespan = 0.0f;

	//Clear our scratch table+
	for (i = 0; i < numMachines; i++)
		scratch[scratchOffset + (i * numParticles) + threadIdx.x] = 0.0f;

	for (i = 0; i < numTasks; i++)
	{
		matchingVal = matching[swarmOffset + (i * numParticles) + threadIdx.x];
		scratch[scratchOffset + (int) (floorf(matchingVal) * numParticles) + threadIdx.x] += tex2D(texETCMatrix, matchingVal, (float) i);
		val = scratch[scratchOffset + (int) (floorf(matchingVal) * numParticles) + threadIdx.x];

		if (val > makespan)
			makespan = val;
	}	

	return makespan;
}

__global__ void UpdateFitness(int numSwarms, int numParticles, int numTasks, int numMachines, float *position, float *scratch, float *fitness)
{
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

	//If we have enough 
	if (threadID < __mul24(numSwarms, numParticles))
		fitness[threadID] = CalcMakespan(numParticles, numTasks, numMachines, position, scratch);
}

/* UpdateBests
 * 
 * Updates both the particle-bests and swarm-best values.
 * Each block must contains enough threads to handle each particle in a swarm.
 * Shared memory requirements are the number of particles in a swarm * 2
 */
__global__ void UpdateBests(int numSwarms, int numParticles, int numTasks, float *pBest, float *pBestPositions, float *gBest, float *gBestPositions,
							float *position, float *fitness)
{
	extern __shared__ float fitnessValues[];
	__shared__ float *indexValues;
	int i;
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int updateFitness;
	int gBestIndex;
	int tidNumTasks;

	indexValues = &fitnessValues[blockDim.x];

	tidNumTasks = threadID * numTasks;

	//Push the fitness values for this swarm into shared memory
	if (threadIdx.x < numParticles)
	{	
		fitnessValues[threadIdx.x] = fitness[threadID];
		indexValues[threadIdx.x] = tidNumTasks;
	}

	//Each thread determines if they need to update their own pbest value.
	//If so, each thread updates the pBest and pBestPosition for their own data.
	if (fitnessValues[threadIdx.x] < pBest[threadID])
	{
		pBest[threadID] = fitnessValues[threadIdx.x];
		
		for (i = 0; i < numTasks; i++)
		{
			pBestPositions[threadID * numTasks + i] = position[tidNumTasks + i];
		}
	}

	__syncthreads();

	//Parallel reduction to find best fitness amongst threads in swarm
	//We do this reduction in shared memory.
	for (i = blockDim.x / 2; i > 0; i >>= 1)
	{
		if (threadIdx.x < i)
		{
			if (fitnessValues[threadIdx.x] > fitnessValues[threadIdx.x + i])
			{				
				fitnessValues[threadIdx.x] = fitnessValues[threadIdx.x + i];
				indexValues[threadIdx.x] = indexValues[threadIdx.x + i];
			}
		}
		__syncthreads();
	}

	//All threads check if gBest must be updated (we just do this to avoid collaboration)
	//Both the shared and global memory values will be broadcast anyways as each thread is
	//accessing the same value, so the performance loss will be minimal.
	updateFitness = 0;	
	__syncthreads();

	if (fitnessValues[0] < gBest[blockIdx.x])
	{		
		updateFitness = 1;	
	}

	//Update gBest and gBestPosition by using all threads in a for loop if necessary
	if (updateFitness)
	{
		for (i = threadIdx.x; i < numTasks; i += blockDim.x)
		{
			gBestPositions[blockIdx.x * numTasks + i] = pBestPositions[(int) indexValues[0] + i];			
		}

		if (threadIdx.x == 0)
		{
			//printf("Found global  best value: %f\n", fitnessValues[0]);
			gBest[blockIdx.x] = fitnessValues[0];
		}
	}
}

/* InitializeParticles
 *
 * Initializes the position and velocity of the particles. Each thread is resposible
 * for a single dimension of a single particle.
 */
__global__ void InitializeParticles(int numSwarms, int numParticles, int numTasks, int numMachines, float *gBests, float *pBests, float *position, 
									float *velocity, float *randNums)
{
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float myRand1, myRand2;
	int randOffset;
	int totalParticles = __mul24(numSwarms, numParticles);

	if (threadID < __mul24(totalParticles, numTasks))
	{
		randOffset = __mul24(totalParticles, numTasks);
		myRand1 = randNums[threadID];
		myRand2 = randNums[threadID + randOffset];
		position[threadID] = (numMachines - 1) * myRand1;	
		velocity[threadID] = (numMachines >> 1) * myRand2;
		
		if (threadID < totalParticles)
		{
			pBests[threadID] = 99999999.99f;

			if (threadID < numSwarms)
			{
				gBests[threadID] = 999999999.99f;
			}
		}
	}
}


__global__ void SwapBestParticles(int numSwarms, int numParticles, int numTasks, int numToSwap, int *bestSwapIndices, int *worstSwapIndices, 
								  float *position, float *velocity, float *pBest, float *pBestPosition)
{
	int bestIndex, worstIndex;
	int neighbor;
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float tempPosition, tempVelocity, tempPBestPosition;
	int mySwarm, mySwapIndex, neighborSwapIndex;
	int mySwapIndicesBase, neighborSwapIndicesBase;
	int myDimension;

	if (threadID < numSwarms * numToSwap * numTasks)
	{	
		//First, figure out what swarm we are covering and who our neighbor is...
		mySwarm = threadID / (numToSwap * numTasks);
		neighbor = mySwarm < numSwarms - 1? mySwarm + 1 : 0;

		//Now, figure out what our starting index is into the swap indices (numToSwap records for each swarm)
		mySwapIndicesBase = mySwarm * (numToSwap);
		neighborSwapIndicesBase = neighbor * (numToSwap);		

		//Now let's figure out which actual swap within this swap we're responsible for as there's numToSwap choices!
		//And, while we're at it, figure out what dimension we're covering.
		mySwapIndex = (threadID / numTasks) % numToSwap;
		neighborSwapIndex = neighbor > 0 ? mySwapIndex : (threadID / numTasks) % numToSwap;
		myDimension = (threadID % numTasks);
		//printf("thread id %d dimension is %d\n", threadID, myDimension);

		//printf("t %d has myswap %d, neig %d with mySwapIndex of %d\n", threadID, mySwapIndicesBase, neighborSwapIndicesBase, mySwapIndex);

		//Finally let's get our indices!!
		bestIndex = (mySwarm * numParticles * numTasks) + (bestSwapIndices[mySwapIndicesBase + mySwapIndex] * numTasks) + myDimension;
		worstIndex = (neighbor * numParticles * numTasks) + (worstSwapIndices[neighborSwapIndicesBase + neighborSwapIndex] * numTasks) + myDimension;

//printf("Thread %d is choosing swaps from %d for best and %d for worst\n", threadID, mySwapIndicesBase + mySwapIndex, neighborSwapIndicesBase + neighborSwapIndex);
//printf("Thread %d will be taking from %d and putting in %d\n", threadID, bestIndex + myDimension, worstIndex + myDimension);


		//Store the best positions temporarily.
		tempPosition = position[bestIndex];
		tempVelocity = velocity[bestIndex];
		tempPBestPosition = pBestPosition[bestIndex];

		//Swap the other swarm's worst into our best
		position[bestIndex] = position[worstIndex];
		velocity[bestIndex] = velocity[worstIndex];
		pBestPosition[bestIndex] = pBestPosition[worstIndex];

		//Finally swap our best values into the other swarm's worst
		position[worstIndex] = tempPosition;
		velocity[worstIndex] = tempVelocity;
		pBestPosition[worstIndex] = tempPBestPosition;

		//Update the pBest value...
		if (threadID < numSwarms * numToSwap)
		{
			mySwarm = threadID / numToSwap;
			neighbor = mySwarm < numSwarms - 1 ? mySwarm + 1 : 0;
			mySwapIndex = threadID % numToSwap;

			bestIndex = mySwarm * numParticles + bestSwapIndices[mySwarm * numToSwap + mySwapIndex];
			worstIndex = neighbor * numParticles + worstSwapIndices[neighbor * numToSwap + mySwapIndex];

			//printf("Thread %d choosing from swap index %d for best and %d for worst\n", threadID, mySwarm * numToSwap + mySwapIndex, neighbor * numToSwap + mySwapIndex);

			tempPosition = pBest[bestIndex];
			pBest[bestIndex] = pBest[worstIndex];
			pBest[worstIndex] = tempPosition;
		}
		
	}
}

/* GenerateSwapIndices
 *
 * Generates the swap indices for swapping particles. Finds the best numToSwap and
 * the worst numToSwap particles from each swarm and records the values.
 *
 * @BLOCKDIM - Requires numParticles particles per thread block.
 * @SHAREDMEM - Requires numParticles * 5 + numToSwap * 2 elements of shared memory.
 */
__global__ void GenerateSwapIndices(int numSwarms, int numParticles, int numToSwap, float *fitness, int *bestSwapIndices, int *worstSwapIndices)
{
	extern __shared__ float sharedFitnessOriginal[];
	__shared__ float *sharedFitnessBest, *sharedFitnessWorst;
	__shared__ float *sharedIndicesBest, *sharedIndicesWorst;
	__shared__ float *sharedTempIndicesBest, *sharedTempIndicesWorst;
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int i, j;

	sharedFitnessBest = &sharedFitnessOriginal[blockDim.x];	
	sharedFitnessWorst = &sharedFitnessBest[blockDim.x];
	sharedTempIndicesBest = &sharedFitnessWorst[blockDim.x];
	sharedTempIndicesWorst = &sharedTempIndicesBest[blockDim.x];	
	sharedIndicesBest = &sharedTempIndicesWorst[blockDim.x];
	sharedIndicesWorst = &sharedIndicesBest[numToSwap];	

	//Push the fitness values for this swarm into shared memory
	if (threadIdx.x < numParticles)
	{	
		sharedFitnessOriginal[threadIdx.x] = fitness[threadID];
		sharedFitnessBest[threadIdx.x] = sharedFitnessOriginal[threadIdx.x];
		sharedFitnessWorst[threadIdx.x] = sharedFitnessBest[threadIdx.x];
		sharedTempIndicesBest[threadIdx.x] = threadIdx.x;
		sharedTempIndicesWorst[threadIdx.x] = threadIdx.x;
	}

	//Main loop to find the best/worst particles.
	for (i = 0; i < numToSwap; i++)
	{		
		for (j = blockDim.x / 2; j > 0; j >>= 1)
		{			
			if (threadIdx.x < j)
			{
				if (sharedFitnessBest[threadIdx.x] == -1 ||
					(sharedFitnessBest[threadIdx.x] > sharedFitnessBest[threadIdx.x + j] && sharedFitnessBest[threadIdx.x + j] != -1))
				{
					//printf("\t[[BEST]]Thread %d grabbing data %f to replace %f from (%d, %d)\n", threadIdx.x, sharedFitnessBest[threadIdx.x + j], sharedFitnessBest[threadIdx.x],
																						//threadIdx.x + j, threadIdx.x);
					sharedFitnessBest[threadIdx.x] = sharedFitnessBest[threadIdx.x + j];
					sharedTempIndicesBest[threadIdx.x] = sharedTempIndicesBest[threadIdx.x + j];
				}

				if (sharedFitnessWorst[threadIdx.x] == -1 ||
					(sharedFitnessWorst[threadIdx.x] < sharedFitnessWorst[threadIdx.x + j] && sharedFitnessWorst[threadIdx.x + j] != -1))
				{				
					//printf("\t[[WORST]]Thread %d grabbing data %f to replace %f from (%d, %d)\n", threadIdx.x, sharedFitnessWorst[threadIdx.x + j], sharedFitnessWorst[threadIdx.x],
																						//threadIdx.x + j, threadIdx.x);
					sharedFitnessWorst[threadIdx.x] = sharedFitnessWorst[threadIdx.x + j];
					sharedTempIndicesWorst[threadIdx.x] = sharedTempIndicesWorst[threadIdx.x + j];
				}
			}
			
			__syncthreads();
		}
		
		//Replace the index with -1 in the originals
		if (threadIdx.x == 0)
		{
			sharedIndicesBest[i] = sharedTempIndicesBest[0];
			sharedIndicesWorst[i] = sharedTempIndicesWorst[0];

					//printf("We found the best %d value for swarm %d as %f at index %d\n", i, blockIdx.x, sharedFitnessBest[0], __float2int_rn(sharedIndicesBest[i]));
		//printf("We found the worst %d value for swarm %d as %f at index %f\n", i, blockIdx.x, sharedFitnessWorst[0], __float2int_rn(sharedIndicesWorst[i]));

			bestSwapIndices[blockIdx.x * numToSwap + i] = __float2int_rn(sharedIndicesBest[i]);
			worstSwapIndices[blockIdx.x * numToSwap + i] = __float2int_rn(sharedIndicesWorst[i]);

			//printf("Wrote out best value as %d to index %d\n", bestSwapIndices[blockDim.x * numToSwap + i], blockIdx.x * numToSwap + i);
		}

		if (threadIdx.x == 0)
		{
			sharedFitnessOriginal[__float2int_rz(sharedIndicesBest[i])] = -1.0f;
			sharedFitnessOriginal[__float2int_rz(sharedIndicesWorst[i])] = -1.0f;
		}

		__syncthreads();

		sharedFitnessBest[threadIdx.x] = sharedFitnessOriginal[threadIdx.x];
		sharedFitnessWorst[threadIdx.x] = sharedFitnessOriginal[threadIdx.x];

		sharedTempIndicesBest[threadIdx.x] = threadIdx.x;
		sharedTempIndicesWorst[threadIdx.x] = threadIdx.x;

		__syncthreads();

	}//for...

}

/* ClampVelocity
 *
 * Clamps a particle's velocity along one dimension to the
 * prescribed level (currently: half of the number of machines)
 */
__device__ float ClampVelocity(int numMachines, float velocity)
{
	float clamp = 0.5f * numMachines;

	if (velocity > clamp)
		velocity = clamp;
	else if (velocity < -clamp)
		velocity = -clamp;

	return velocity;
}

/* ClampPosition
 *
 * Clamps a particle's position along one dimension ensuring
 * it cannot go outside the valid bounds of the solution space: [0, numMachines - 1].
 */
__device__ float ClampPosition(int numMachines, float position)
{
	if (position < 0.0f)
		position = 0.0f;
	else if (position > numMachines - 1)
		position = (float) numMachines - 1;

	return position;
}

__global__ void UpdateVelocityAndPosition(int numSwarms, int numParticles, int numMachines, int numTasks, float *velocity, float *position, 
										  float *pBestPosition, float *gBestPosition, float *rands, ArgStruct args)
{
	int threadID = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float newVel;
	float currPos;
	int randOffset;
	int totalParticles = numSwarms * numParticles;
	int gBestOffset;

	//Each thread is responsible for updating one dimension of one particle's 
	if (threadID < __mul24(totalParticles, numTasks))
	{
		//Two separate random numbers for every dimension for each particle each iteration.
		randOffset = threadID * 2;

		//The swarm this particle belongs to simply the number of threads handling each swarm (numParticles * numTasks)
		//divided by this thread's threadID.
		gBestOffset = (threadID / (numParticles * numTasks)) * numTasks;
		gBestOffset += threadID % numTasks;

		currPos = position[threadID];
		newVel = velocity[threadID];

		newVel *= args.x;		
		newVel += args.z * rands[randOffset] * (pBestPosition[threadID] - currPos);
		newVel += args.w * rands[randOffset + 1] * (gBestPosition[gBestOffset] - currPos);	

		//Write out our velocity
		newVel = ClampVelocity(numMachines, newVel);
		velocity[threadID] = newVel;

		//Update the position
		currPos += newVel;
		currPos = ClampPosition(numMachines, currPos);
		position[threadID] = currPos;
	}
}

void InitTexture()
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	texETCMatrix.normalized = false;
	texETCMatrix.filterMode = hipFilterModePoint;
	texETCMatrix.addressMode[0] = hipAddressModeClamp;
    texETCMatrix.addressMode[1] = hipAddressModeClamp;

	hipMallocArray(&cuArray, &channelDesc, GetNumMachines(), GetNumTasks());
	hipMemcpyToArray(cuArray, 0, 0, hETCMatrix, sizeof(float) * GetNumMachines() * GetNumTasks(), hipMemcpyHostToDevice);
	hipBindTextureToArray(texETCMatrix, cuArray, channelDesc);
}

void ClearTexture()
{
	hipUnbindTexture(texETCMatrix);
	hipFreeArray(cuArray);
}

/* GenerateRandomNumbers
 *
 * Generates as many iterations worth of random numbers such that the overall size
 * of random numbers generated does not exceed maxGen.
 *
 * Returns the number of iterations covered by this generation of random numbers.
 */
int GenerateRandomNumbers(int totalParticles, int numTasks, int iterationsRemaining, float *dRand)
{
	int totalRandsPerIter;
	int numIterationsCovered;

	totalRandsPerIter = totalParticles * numTasks * 2;

	numIterationsCovered = MAX_RAND_GEN / (totalRandsPerIter * sizeof(float));
	numIterationsCovered = numIterationsCovered == 0 ? 1 : numIterationsCovered;
	numIterationsCovered = numIterationsCovered > iterationsRemaining ? iterationsRemaining : numIterationsCovered;

	GenRandsGPU(totalRandsPerIter * numIterationsCovered, dRand);

	return numIterationsCovered;
}

float *MRPSODriver(RunConfiguration *run)
{
	int i, j;
	int threadsPerBlock, threadsPerBlockSwap, numBlocks, numBlocksFitness, numBlocksSwap;
	int fitnessRequired;
	int totalComponents;
	int numMachines, numTasks;
	ArgStruct args;
	float *gBests = NULL;
	float *gBestsTemp;
	float minVal;
	int swapSharedMem;
	int itersOfRands;
	int itersOfRandsLeft;
	int dRandsOffset;

#ifdef KERNEL_TIMING
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif

#ifdef RECORD_VALUES
	gBests = (float *) malloc(run->numIterations * sizeof(float));
	gBestsTemp = (float *) malloc(run->numSwarms * sizeof(float));
#endif

	numMachines = GetNumMachines();
	numTasks = GetNumTasks();
	threadsPerBlock = run->threadsPerBlock;
	totalComponents = run->numSwarms * run->numParticles * numTasks;
	fitnessRequired = run->numSwarms * run->numParticles;
	args.x = run->w;
	args.y = run->wDecay;
	args.z = run->c1;
	args.w = run->c2;

	numMachines = GetNumMachines();
	numTasks = GetNumTasks();

	swapSharedMem = (run->numParticles * 5 + run->numParticlesToSwap * 2) * sizeof(float);

	numBlocks = CalcNumBlocks(totalComponents, threadsPerBlock);
	numBlocksFitness = CalcNumBlocks(fitnessRequired, 128);
	threadsPerBlockSwap = 64;
	numBlocksSwap = CalcNumBlocks(run->numSwarms * run->numParticlesToSwap, threadsPerBlockSwap);

	//Generate the random numbers we need for the initialization...
	InitRandsGPU();
	GenRandsGPU(run->numSwarms * run->numParticles * numTasks * 2, dRands);

#ifdef KERNEL_TIMING
	hipEventRecord(start, 0);
#endif

	//Initialize our particles.
	InitializeParticles<<<numBlocks, threadsPerBlock>>>(run->numSwarms, run->numParticles, numTasks, numMachines, dGBest, dPBest, dPosition, dVelocity, dRands);

#ifdef KERNEL_TIMING
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	initTime += elapsedTime;
#endif

	itersOfRandsLeft = 0;

	//Run MRPSO GPU for the given number of iterations.
	for (i = 1; i <= run->numIterations; i++)
	{
		//If we need to generate more random numbers then do so now...
		if (!itersOfRandsLeft)
		{
			itersOfRands = GenerateRandomNumbers(run->numSwarms * run->numParticles, numTasks, run->numIterations - i + 1, dRands);
			itersOfRandsLeft = itersOfRands;
		}

		//Update the Fitness
		UpdateFitness<<<numBlocksFitness, 128>>>(run->numSwarms, run->numParticles, numTasks, numMachines, dPosition, dScratch, dFitness);

		//Update the local and swarm best positions
		UpdateBests<<<run->numSwarms, run->numParticles, run->numParticles * 2 * sizeof(float)>>>(run->numSwarms, run->numParticles, numTasks, dPBest, 
																							      dPBestPosition, dGBest, dGBestPosition,
														                                          dPosition, dFitness);
#ifdef RECORD_VALUES
		hipDeviceSynchronize();
		hipMemcpy(gBestsTemp, dGBest, run->numSwarms * sizeof(float), hipMemcpyDeviceToHost);

		minVal = gBestsTemp[0];

		//Find the minimal gbest value
		for (j = 1; j < run->numSwarms; j++)
		{
			if (gBestsTemp[j] < minVal)
				minVal = gBestsTemp[j];
		}

		gBests[i - 1] = minVal;
#endif

		//REMINDER: The problem lies in the random number use after a certain number of iterations.
		//Update the Position and Velocity
		dRandsOffset = (itersOfRands - itersOfRandsLeft) * run->numSwarms * run->numParticles * numTasks * 2;
		UpdateVelocityAndPosition<<<numBlocks, threadsPerBlock>>>(run->numSwarms, run->numParticles, numMachines, numTasks, 
																  dVelocity, dPosition, dPBestPosition, dGBestPosition, &dRands[dRandsOffset], args);	

		if (args.x > 0.0f)
			args.x *= run->wDecay;

		if (i % run->iterationsBeforeSwap == 0)
		{
			//Build up the swap indices for each swarm
			GenerateSwapIndices<<<run->numSwarms, run->numParticles, swapSharedMem>>>(run->numSwarms, run->numParticles, run->numParticlesToSwap, 
			                                                                          dFitness, dBestSwapIndices, dWorstSwapIndices);

			//Swap particles between swarms
			SwapBestParticles<<<numBlocksSwap, threadsPerBlockSwap>>>(run->numSwarms, run->numParticles, numTasks, run->numParticlesToSwap, dBestSwapIndices, 
																	  dWorstSwapIndices, dPosition, dVelocity, dPBest, dPBestPosition);
		}

		itersOfRandsLeft--;
	}

	FreeRandsGPU();

	return gBests;
}

