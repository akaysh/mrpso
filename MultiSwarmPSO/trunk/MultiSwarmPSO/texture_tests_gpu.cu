#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "tests.h"
#include "helper.h"

texture<float, 2, hipReadModeElementType> texETCMatrix;

__device__ int GetDiscreteCoordT(float val)
{
	return  floorf(val);
}

__global__ void TestTexture(int numTasks, int numMachines, float *outVals)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx < numTasks * numMachines)
		outVals[idx] = tex2D(texETCMatrix, threadIdx.x, blockIdx.x);
}

__global__ void TestRandTexture(float *dVals, float *dOut, int numTasks, int numMachines)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadID < numTasks)
		dOut[threadID] = tex2D(texETCMatrix, dVals[threadID], (float) (threadID));
}

int TestTextureReads()
{
	int i;
	int passed = 1;
	hipArray *cuArray;
	float *dOut;
	float *gpuETCMatrix;

	printf("\tRunning Texture Read Test...\n");
	
	BuildMachineList("machines100.txt");
	BuildTaskList("tasks1000.txt");
	GenerateETCMatrix();

	gpuETCMatrix = (float *) malloc(GetNumMachines() * GetNumTasks() * sizeof(float));
	hipMalloc((void **)&dOut, GetNumMachines() * GetNumTasks() * sizeof(float));

	hipMallocArray(&cuArray, &texETCMatrix.channelDesc, GetNumMachines(), GetNumTasks());
	hipMemcpyToArray(cuArray, 0, 0, hETCMatrix, sizeof(float)*GetNumMachines() *GetNumTasks(), hipMemcpyHostToDevice);
	hipBindTextureToArray(texETCMatrix, cuArray);

	texETCMatrix.normalized = false;
	texETCMatrix.filterMode = hipFilterModePoint;

	TestTexture<<<1000, 100>>>(GetNumTasks(), GetNumMachines(), dOut);
	hipDeviceSynchronize();

	hipMemcpy(gpuETCMatrix, dOut, sizeof(float)*GetNumMachines() *GetNumTasks(), hipMemcpyDeviceToHost);

	for (i = 0; i < GetNumTasks() * GetNumMachines(); i++)
	{
		if (gpuETCMatrix[i] - hETCMatrix[i] > ACCEPTED_DELTA)
		{
			printf("\t[ERROR] - GPU ETC Matrix was: %f (expected: %f)\n", gpuETCMatrix[i], hETCMatrix[i]);
			passed = 0;
		}
	}

	PrintTestResults(passed);

	free(gpuETCMatrix);
	FreeCPUMemory();
	hipFree(dOut);
	hipFreeArray(cuArray);	
	
	return passed;
}

int TestTextureReadsRandom()
{
	int i;
	int passed = 1;
	hipArray *cuArray;
	float *dOut;
	float *gpuETCMatrix;
	float *hMatching, *dMatching;
	float *cpuOut, *gpuOut;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	int threadsPerBlock, numBlocks;

	threadsPerBlock = 64;

	printf("\tRunning Texture Read Random Test...\n");
	
	BuildMachineList("machines8.txt");
	BuildTaskList("tasks80.txt");
	GenerateETCMatrix();

	srand((unsigned int) time(NULL));

	gpuETCMatrix = (float *) malloc(GetNumMachines() * GetNumTasks() * sizeof(float));
	hMatching = (float *) malloc(GetNumTasks() * sizeof(float));
	cpuOut = (float *) malloc(GetNumTasks() * sizeof(float));
	gpuOut = (float *) malloc(GetNumTasks() * sizeof(float));

	hipMalloc((void **)&dOut, GetNumMachines() * GetNumTasks() * sizeof(float));
	hipMalloc((void **)&dMatching, GetNumTasks() * sizeof(float));
	hipMalloc((void **)&dOut, GetNumTasks() * sizeof(float));

	texETCMatrix.normalized = false;
	texETCMatrix.filterMode = hipFilterModePoint;
	texETCMatrix.addressMode[0] = hipAddressModeClamp;
    texETCMatrix.addressMode[1] = hipAddressModeClamp;

	hipMallocArray(&cuArray, &channelDesc, GetNumMachines(), GetNumTasks());
	hipMemcpyToArray(cuArray, 0, 0, hETCMatrix, sizeof(float)*GetNumMachines() *GetNumTasks(), hipMemcpyHostToDevice);
	hipBindTextureToArray(texETCMatrix, cuArray, channelDesc);

	for (i = 0; i < GetNumTasks(); i++)
		hMatching[i] = (float) (rand() % ((GetNumMachines() - 1) * 100)) / 100.0f;

	hipMemcpy(dMatching, hMatching, GetNumTasks() * sizeof(float), hipMemcpyHostToDevice);

	numBlocks = CalcNumBlocks(GetNumTasks(), threadsPerBlock);

	TestRandTexture<<<numBlocks, threadsPerBlock>>>(dMatching, dOut, GetNumTasks(), GetNumMachines());
	hipDeviceSynchronize();

	hipMemcpy(gpuOut, dOut, GetNumTasks() * sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < GetNumTasks(); i++)
		cpuOut[i] = hETCMatrix[(i * GetNumMachines()) + DiscreteCoord(hMatching[i])];

	for (i = 0; i < GetNumTasks(); i++)
	{
		if (abs(gpuOut[i] - cpuOut[i]) > ACCEPTED_DELTA)
		{
			printf("\t[ERROR] - %d GPU ETC Matrix was: %f (expected: %f)\n", i, gpuOut[i], cpuOut[i]);
			printf("\t\tOriginal matching value used: %f\n", hMatching[i]);
			passed = 0;
		}
	}

	PrintTestResults(passed);

	free(gpuETCMatrix);
	FreeCPUMemory();
	hipFree(dOut);
	hipFreeArray(cuArray);	
	
	return passed;
}

void RunGPUTextureTests()
{
	int passed = 1;

	printf("\nStarting GPU Texture tests...\n\n");

	passed &= TestTextureReads();
	passed &= TestTextureReadsRandom();

	if (passed)
		printf("[PASSED] All texture tests passed!\n\n");
	else
		printf("[FAILED] Texture tests failed!\n\n");
}
