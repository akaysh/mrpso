#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "tests.h"
#include "helper.h"
#include "gpu_pso.h"

int floatcomp(const void* elem1, const void* elem2)
{
    if(*(const float*)elem1 < *(const float*)elem2)
        return -1;
    return *(const float*)elem1 > *(const float*)elem2;
}

void GenBestWorst(int count, int numToSwap, float *fitness, int *bestIndices, int *worstIndices)
{
	int i, j;
	int currBestIndex, currWorstIndex;
	float currBestValue, currWorstValue;
	float *fitnessTemp;

	fitnessTemp = (float *) malloc(count * sizeof(float));

	memcpy(fitnessTemp, fitness, count * sizeof(float));

	for (i = 0; i < numToSwap; i++)
	{
		currBestValue = 999999999.9f;
		currWorstValue = -1.0f;

		//Find the best and worst values.
		for (j = 0; j < count; j++)
		{
			if (fitnessTemp[j] != -1.0f && fitnessTemp[j] < currBestValue)
			{
				currBestValue = fitnessTemp[j];
				currBestIndex = j;
			}

			if (fitnessTemp[j] != -1.0f && fitnessTemp[j] > currWorstValue)
			{
				currWorstValue = fitnessTemp[j];
				currWorstIndex = j;
			}
		}

		bestIndices[i] = currBestIndex;
		fitnessTemp[currBestIndex] = -1;

		worstIndices[i] = currWorstIndex;
		fitnessTemp[currWorstIndex] = -1;
	}

	free(fitnessTemp);
}

int TestGenerateSwapIndices()
{
	int passed = 1;
	int found;
	int i, j, k;
	float *hFitness, *dFitness;
	int *hBestSwapIndices, *dBestSwapIndices, *cpuBestSwapIndices;
	int *hWorstSwapIndices, *dWorstSwapIndices, *cpuWorstSwapIndices;
	int numSwarms, numParticles;
	int numToSwap;

	numSwarms = 200;
	numParticles = 64;
	numToSwap = 10;

	printf("\tRunning GPU swap index generation test...\n");

	srand((unsigned int) time(NULL));

	hFitness = (float *) malloc(numSwarms * numParticles * sizeof(float));
	hBestSwapIndices = (int *) malloc(numSwarms * numToSwap * sizeof(int));
	cpuBestSwapIndices = (int *) malloc(numSwarms * numToSwap * sizeof(int));
	hWorstSwapIndices = (int *) malloc(numSwarms * numToSwap * sizeof(int));
	cpuWorstSwapIndices = (int *) malloc(numSwarms * numToSwap * sizeof(int));

	hipMalloc((void **) &dFitness, numSwarms * numParticles * sizeof(float));
	hipMalloc((void **) &dBestSwapIndices, numSwarms * numToSwap * sizeof(int));
	hipMalloc((void **) &dWorstSwapIndices, numSwarms * numToSwap * sizeof(int));

	//Randomly generate our fitness data
	for (i = 0; i < numSwarms * numParticles; i++)
	{
		hFitness[i] = (float) (rand() % 100000 + rand() % 100000 + rand() % 1000 + rand() % 10 + rand() % 10);
	}

	//Push the fitness data to the GPU
	hipMemcpy(dFitness, hFitness, numSwarms * numParticles * sizeof(float), hipMemcpyHostToDevice);

	//Compute the sequential reference solution.
	for (i = 0; i < numSwarms; i++)
	{
		GenBestWorst(numParticles, numToSwap, &hFitness[i * numParticles], &cpuBestSwapIndices[i * numToSwap], &cpuWorstSwapIndices[i * numToSwap]);
	}

	//Compute the GPU solution
	GenerateSwapIndices<<<numSwarms, numParticles, (numParticles * 5 + numToSwap * 2) * sizeof(float)>>>(numSwarms, numParticles, numToSwap, 
		                                                                                                 dFitness, dBestSwapIndices, dWorstSwapIndices);
	hipDeviceSynchronize();

	hipMemcpy(hBestSwapIndices, dBestSwapIndices, numSwarms * numToSwap * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hWorstSwapIndices, dWorstSwapIndices, numSwarms * numToSwap * sizeof(int), hipMemcpyDeviceToHost);

	//Confirm the results...
	for (i = 0; i < numSwarms; i++)
	{
		for (j = 0; j < numToSwap; j++)
		{
			found = 0;

			//Search for this swap index...
			for (k = 0; k < numToSwap && !found; k++)
			{
				if (hBestSwapIndices[i * numToSwap + j] == cpuBestSwapIndices[i * numToSwap + k])
					found = 1;
			}

			if (!found)
			{
				printf("\t[ERROR] - GPU best swap #%d for swarm %d was: %d with fitness %f (expected: %d with fitness %f)\n", j, i, 
					   hBestSwapIndices[i * numToSwap + j], hFitness[i * numParticles + hBestSwapIndices[i * numToSwap + j]], cpuBestSwapIndices[i * numToSwap + j],
					   hFitness[i * numParticles + cpuBestSwapIndices[i * numToSwap + j]]);
				passed = 0;
			}

			found = false;

			//Search for this swap index...
			for (k = 0; k < numToSwap && !found; k++)
			{
				if (hWorstSwapIndices[i * numToSwap + j] == cpuWorstSwapIndices[i * numToSwap + k])
					found = 1;
			}

			if (!found)
			{
				printf("\t[ERROR] - GPU worst swap #%d for swarm %d was: %d (expected: %d)\n", j, i, hWorstSwapIndices[i * numToSwap + j], cpuWorstSwapIndices[i * numToSwap + j]);
				passed = 0;
			}
		}
	}

	PrintTestResults(passed);

	free(hFitness);
	free(hBestSwapIndices);
	free(hWorstSwapIndices);
	free(cpuBestSwapIndices);
	free(cpuWorstSwapIndices);

	hipFree(dFitness);
	hipFree(dBestSwapIndices);
	hipFree(dWorstSwapIndices);

	return passed;
}

int TestSwapParticles()
{
	int i, j, k, mySwarmOffset, previousSwarmValue, neighborSwarmValue;
	int passed = 1;
	Particle *particles;
	float *hPosition, *dPosition, *hVelocity, *dVelocity;
	float *hPBest, *dPBest, *cpuPBest, *hPBestPosition, *dPBestPosition, *cpuPBestPosition;
	int *bestListing;
	int *worstListing;
	int *dBestSwapIndices, *dWorstSwapIndices;
	float *fitnesses;
	int numParticles;
	int numToSwap;
	int numSwarms;
	int numTasks;
	float currFitness;
	int swapIndex;
	int index;
	int threadsPerBlock, numBlocks;	

	numParticles = 128;
	numToSwap = 20;
	numSwarms = 30;
	numTasks = 1000;

	printf("\tRunning particle swap test...\n");

	hPosition = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	hVelocity = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	hPBest = (float *) malloc(numParticles * numSwarms * sizeof(float));
	cpuPBest = (float *) malloc(numParticles * numSwarms * sizeof(float));
	hPBestPosition = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	cpuPBestPosition = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	bestListing = (int *) malloc(numToSwap * numSwarms * sizeof(int));
	worstListing = (int *) malloc(numToSwap * numSwarms * sizeof(int));
	fitnesses = (float *) malloc(numParticles * numSwarms * sizeof(float));

	hipMalloc((void **) &dPosition, numParticles * numSwarms * numTasks * sizeof(float));
	hipMalloc((void **) &dVelocity, numParticles * numSwarms * numTasks * sizeof(float));
	hipMalloc((void **) &dBestSwapIndices, numToSwap * numSwarms * sizeof(int));
	hipMalloc((void **) &dWorstSwapIndices, numToSwap * numSwarms * sizeof(int));
	hipMalloc((void **) &dPBest, numParticles * numSwarms * sizeof(float));
	hipMalloc((void **) &dPBestPosition, numParticles * numSwarms * numTasks * sizeof(float));

	srand((unsigned int) time(NULL));

	//Initialize our Particles
	particles = (Particle *) malloc(numParticles * numSwarms * sizeof(Particle));
	
	for (i = 0; i < numParticles * numSwarms; i++)
	{
		fitnesses[i] = (float) (rand() % 1000000000) + rand() % 1000223 + rand() % (rand() % 100000000) + rand() % (rand() % 100000);
		particles[i].fitness = fitnesses[i];
	}

	//Locate the top numToSwap and worst numToSwap Particles in each swarm by qsorting
	//the fitnesses for each swarm and dumping them into the relevant best/worst listing.
	for (i = 0; i < numSwarms; i++)
	{
		qsort(&fitnesses[numParticles * i], numParticles, sizeof(float), floatcomp);

		index = i * numToSwap;

		for (j = 0; j < numToSwap; j++)
		{
			currFitness = fitnesses[i * numParticles + j];			

			//Search for this fitness value in the particles to get the 'real' index.
			for (k = 0; k < numParticles; k++)
			{
				if (abs(particles[i * numParticles + k].fitness - currFitness) < ACCEPTED_DELTA) //Then we found it, mark the index.
				{
					bestListing[index++] = k;
					break;
				}
			}
		}

		index = i * numToSwap;

		for (j = numParticles - numToSwap; j < numParticles; j++)
		{
			currFitness = fitnesses[i * numParticles + j];			

			//Search for this fitness value in the particles to get the 'real' index.
			for (k = 0; k < numParticles; k++)
			{
				if (abs(particles[i * numParticles + k].fitness - currFitness) < ACCEPTED_DELTA) //Then we found it, mark the index.
				{
					worstListing[index++] = k;
					break;
				}
			}
		}
	}

	//Generate some simple values that we can track for the position and velocities...
	for (i = 0; i < numSwarms; i++)
	{
		for (j = 0; j < numParticles; j++)
		{
			hPBest[(i * numParticles) + j] = (float) i;

			for (k = 0; k < numTasks; k++)
			{
				hPosition[i * numParticles * numTasks + k * numParticles + j] = (float) i;
				hVelocity[i * numParticles * numTasks + k * numParticles + j] = (float) i;
				hPBestPosition[i * numParticles * numTasks + k * numParticles + j] = (float) i;
			}
		}
	}

	//Copy the memory over to the GPU
	hipMemcpy(dPosition, hPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dVelocity, hVelocity, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dPBestPosition, hPBestPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dPBest, hPBest, numParticles * numSwarms * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dBestSwapIndices, bestListing, numToSwap * numSwarms * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWorstSwapIndices, worstListing, numToSwap * numSwarms * sizeof(int), hipMemcpyHostToDevice);

	threadsPerBlock = 64;
	numBlocks = CalcNumBlocks(numSwarms * numToSwap * numTasks, threadsPerBlock);

	SwapBestParticles<<<numBlocks, threadsPerBlock>>>(numSwarms, numParticles, numTasks, numToSwap, dBestSwapIndices, 
		                                              dWorstSwapIndices, dPosition, dVelocity, dPBest, dPBestPosition);
	hipDeviceSynchronize();

	//Copy the data back
	hipMemcpy(hPosition, dPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hVelocity, dVelocity, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hPBestPosition, dPBestPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hPBest, dPBest, numParticles * numSwarms * sizeof(float), hipMemcpyDeviceToHost);

	//Check that the swaps were performed correctly for each swarm.
	for (i = 0; i < numSwarms; i++)
	{
		mySwarmOffset = i * numParticles * numTasks;

		previousSwarmValue = i != 0 ? i - 1 : numSwarms - 1;
		neighborSwarmValue = i < numSwarms - 1 ? i + 1 : 0;

		//Check that this swarm's 'best' positions, velocity, and pbestpositions have the values of the neighboring swarm.
		//Also we want to ensure that this swarm's 'worst' values have the values of the previous swarm.
		for (j = 0; j < numToSwap; j++)
		{		
			swapIndex = (i * numToSwap) + j;

			//Ensure each dimension of the position, velocity and pBestPosition match the expected values as described above.
			for (k = 0; k < numTasks; k++)
			{
				if (abs(hPosition[mySwarmOffset + (k * numParticles) + bestListing[swapIndex]] - neighborSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU best swap position value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, bestListing[swapIndex], k,
						                          hPosition[mySwarmOffset + (bestListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if (abs(hPosition[mySwarmOffset + (k * numParticles) + worstListing[swapIndex]] - previousSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU worst swap position value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, worstListing[swapIndex], k,
						                          hPosition[mySwarmOffset + (worstListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if (abs(hVelocity[mySwarmOffset + (k * numParticles) + bestListing[swapIndex]] - neighborSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU best swap velocity value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, bestListing[swapIndex], k,
						                          hVelocity[mySwarmOffset + (bestListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if (abs(hVelocity[mySwarmOffset + (k * numParticles) + worstListing[swapIndex]] - previousSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU worst swap velocity value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, worstListing[swapIndex], k,
						                          hVelocity[mySwarmOffset + (worstListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if (abs(hPBestPosition[mySwarmOffset + (k * numParticles) + bestListing[swapIndex]] - neighborSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU best swap PBestPosition value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, bestListing[swapIndex], k,
						                          hPBestPosition[mySwarmOffset + (bestListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if (abs(hPBestPosition[mySwarmOffset + (k * numParticles) + worstListing[swapIndex]] - previousSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU best swap PBestPosition value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, worstListing[swapIndex], k,
						                          hPBestPosition[mySwarmOffset + (worstListing[swapIndex] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}
			}
		}
	}	

	free(hPosition);
	free(hVelocity);
	free(bestListing);
	free(worstListing);
	free(fitnesses);
	free(particles);
	free(hPBest);
	free(cpuPBest);
	free(cpuPBestPosition);
	free(hPBestPosition);

	hipFree(dPosition);
	hipFree(dVelocity);
	hipFree(dBestSwapIndices);
	hipFree(dWorstSwapIndices);
	hipFree(dPBest);
	hipFree(dPBestPosition);

	PrintTestResults(passed);

	return passed;
}

void RunSwapTests()
{
	int passed = 1;

	printf("\nStarting GPU swap tests...\n\n");

	passed &= TestSwapParticles();
	passed &= TestGenerateSwapIndices();

	if (passed)
		printf("[PASSED] All swap tests passed!\n\n");
	else
		printf("[FAILED] Swap tests failed!\n\n");
}
