#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "tests.h"
#include "helper.h"
#include "gpu_pso.h"

int floatcomp(const void* elem1, const void* elem2)
{
    if(*(const float*)elem1 < *(const float*)elem2)
        return -1;
    return *(const float*)elem1 > *(const float*)elem2;
}

int TestSwapParticles()
{
	int i, j, k, mySwarmOffset, previousSwarmOffset, neighborSwarmOffset, previousSwarmValue, neighborSwarmValue;
	int passed = 1;
	Particle *particles;
	float *hPosition, *dPosition, *hVelocity, *dVelocity;
	int *bestListing;
	int *worstListing;
	int *dBestSwapIndices, *dWorstSwapIndices;
	float *fitnesses;
	int numParticles;
	int numToSwap;
	int numSwarms;
	int numTasks;
	int numMachines;
	float currFitness;
	int index;
	int threadsPerBlock, numBlocks;	

	numParticles = 5;
	numToSwap = 2;
	numSwarms = 2;
	numTasks = 10;
	numMachines = 4;

	printf("\tRunning particle swap test...\n");

	hPosition = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	hVelocity = (float *) malloc(numParticles * numSwarms * numTasks * sizeof(float));
	bestListing = (int *) malloc(numToSwap * numSwarms * sizeof(int));
	worstListing = (int *) malloc(numToSwap * numSwarms * sizeof(int));
	fitnesses = (float *) malloc(numParticles * numSwarms * sizeof(float));

	hipMalloc((void **) &dPosition, numParticles * numSwarms * numTasks * sizeof(float));
	hipMalloc((void **) &dVelocity, numParticles * numSwarms * numTasks * sizeof(float));
	hipMalloc((void **) &dBestSwapIndices, numToSwap * numSwarms * sizeof(int));
	hipMalloc((void **) &dWorstSwapIndices, numToSwap * numSwarms * sizeof(int));

	srand((unsigned int) time(NULL));

	//Initialize our Particles
	particles = (Particle *) malloc(numParticles * numSwarms * sizeof(Particle));
	
	for (i = 0; i < numParticles * numSwarms; i++)
	{
		fitnesses[i] = (rand() % 1000) + 1;
		particles[i].fitness = fitnesses[i];
	}

	//Locate the top numToSwap and worst numToSwap Particles in each swarm by qsorting
	//the fitnesses for each swarm and dumping them into the relevant best/worst listing.
	for (i = 0; i < numSwarms; i++)
	{
		qsort(&fitnesses[numParticles * i], numParticles, sizeof(float), floatcomp);

		index = i * numToSwap;

		for (j = 0; j < numToSwap; j++)
		{
			currFitness = fitnesses[i * numParticles + j];			

			//Search for this fitness value in the particles to get the 'real' index.
			for (k = 0; k < numParticles; k++)
			{
				if (abs(particles[i * numParticles + k].fitness - currFitness) < ACCEPTED_DELTA) //Then we found it, mark the index.
				{
					bestListing[index++] = k;
					break;
				}
			}
		}

		index = i * numToSwap;

		for (j = numParticles - numToSwap; j < numParticles; j++)
		{
			currFitness = fitnesses[i * numParticles + j];			

			//Search for this fitness value in the particles to get the 'real' index.
			for (k = 0; k < numParticles; k++)
			{
				if (abs(particles[i * numParticles + k].fitness - currFitness) < ACCEPTED_DELTA) //Then we found it, mark the index.
				{
					worstListing[index++] = k;
					break;
				}
			}
		}
	}

	//Generate some simple values that we can track for the position and velocities...
	for (i = 0; i < numSwarms; i++)
	{
		for (j = 0; j < numParticles; j++)
		{
			for (k = 0; k < numTasks; k++)
			{
				hPosition[(i * numParticles * numTasks) + j * numTasks + k] = (float) i;
				hVelocity[(i * numParticles * numTasks) + j * numTasks + k] = (float) i;
			}
		}
	}

	//Copy the memory over to the GPU
	hipMemcpy(dPosition, hPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dVelocity, hVelocity, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dBestSwapIndices, bestListing, numToSwap * numSwarms * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWorstSwapIndices, worstListing, numToSwap * numSwarms * sizeof(int), hipMemcpyHostToDevice);

	threadsPerBlock = 32;
	numBlocks = CalcNumBlocks(numSwarms * numToSwap, threadsPerBlock);

	SwapBestParticles<<<numBlocks, threadsPerBlock>>>(numSwarms, numParticles, numTasks, numToSwap, dBestSwapIndices, dWorstSwapIndices, dPosition, dVelocity);
	hipDeviceSynchronize();

	//Copy the data back
	hipMemcpy(hPosition, dPosition, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(hVelocity, dVelocity, numParticles * numSwarms * numTasks * sizeof(float), hipMemcpyDeviceToHost);

	//Ensure that the correct modified numbers were added.
	for (i = 0; i < numSwarms; i++)
	{
		mySwarmOffset = i * numParticles * numTasks;

		previousSwarmValue = i != 0 ? i - 1 : numSwarms - 1;
		previousSwarmOffset = previousSwarmValue * numParticles * numTasks;

		neighborSwarmValue = i < numSwarms - 1 ? i + 1 : 0;
		neighborSwarmOffset = neighborSwarmValue * numParticles * numTasks;

		//For this swarm ensure our 'best' particles position and velocity values are now equal to our neighboring swarm's values.
		//				 ensure our 'worst' particles position and velocity values are now equal to the "previous" swarm's values.
		for (j = 0; j < numToSwap; j++)
		{
			for (k = 0; k < numTasks; k++)
			{
				if(abs(hPosition[mySwarmOffset + (bestListing[j + (i * numToSwap)] * numTasks) + k] - neighborSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU Position value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, bestListing[j], k,
						                          hPosition[mySwarmOffset + (bestListing[j * (i * numToSwap)] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}

				if(abs(hVelocity[mySwarmOffset + (bestListing[j + (i * numToSwap)] * numTasks) + k] - neighborSwarmValue) > ACCEPTED_DELTA)
				{
					printf("\t[ERROR] - GPU Velocity value for swarm %d, particle %d, element %d was: %f (expected: %d)\n", i, bestListing[j], k,
						                          hVelocity[mySwarmOffset + (bestListing[j * (i * numToSwap)] * numTasks) + k], neighborSwarmValue);
					passed = 0;
				}
			}
		}
	}

	PrintTestResults(passed);

	return passed;
}

void RunSwapTests()
{
	int passed = 1;

	printf("\nStarting GPU swap tests...\n\n");

	passed &= TestSwapParticles();

	if (passed)
		printf("[PASSED] All swap tests passed!\n\n");
	else
		printf("[FAILED] Swap tests failed!\n\n");
}